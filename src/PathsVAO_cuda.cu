#include "hip/hip_runtime.h"

#include "PathsVAO.h"


#define VCOUNT (9)
#define VCOUNT2 (18)


//インターリーブ配列としてのインデックスと
//頂点配列としてのインデックスを区別するため、
//接尾辞を下記にする
//_index  : 頂点配列としてのインデックス
//_iindex : インターリーブ配列としてのインデックス


__host__
__device__
bool isInInterval
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_iindex,
		unsigned int end_iindex,
		unsigned int  vertex_iindex
	)
{
	const unsigned int it = 3;
	if((vertex_iindex < begin_iindex) || (end_iindex <= vertex_iindex))
	{
		return false;
	}
	if((vertex_d[vertex_iindex + it] <= time) && (time < vertex_d[vertex_iindex + VCOUNT2 + it]))
	{
		return true;
	}
	return false;
}

__host__
__device__
void updateTimeVertex
	(
	 float time,
	 const float* const vertex_d,
	 unsigned int begin_iindex,
	 unsigned int end_iindex,
	 unsigned int v_iindex,
	 float* const time_vertex
	)
{
	//各座標を示すインデックス
	const unsigned int ix = 0;
	const unsigned int iy = 1;
	const unsigned int iz = 2;
	const unsigned int it = 3;
	const unsigned int ir = 4;
	const unsigned int ig = 5;
	const unsigned int ib = 6;
	const unsigned int ia = 7;
	
	unsigned int from_iindex = (v_iindex < begin_iindex) ? begin_iindex : v_iindex;
	unsigned int   to_iindex = (v_iindex < begin_iindex) ? begin_iindex : (v_iindex + VCOUNT2);
	from_iindex = (end_iindex <= from_iindex) ? (end_iindex - 1) : from_iindex;
	  to_iindex = (end_iindex <=   to_iindex) ? (end_iindex - 1) :   to_iindex;
	
	const float* const from = &vertex_d[from_iindex];
	const float* const to   = &vertex_d[  to_iindex];
	float from_time = from[it];
	float   to_time =   to[it];

	//内分比
	float ratio_from = 1.0 - ((float)(time - from_time))/((float)(to_time - from_time));
	float ratio_to = 1.0 - ratio_from;
	
	//上の点の設定
	time_vertex[ix] = ratio_from * from[ix] + ratio_to * to[ix];
	time_vertex[iy] = ratio_from * from[iy] + ratio_to * to[iy];
	time_vertex[iz] = ratio_from * from[iz] + ratio_to * to[iz];
	time_vertex[it] = time;
	time_vertex[ir] = ratio_from * from[ir] + ratio_to * to[ir];
	time_vertex[ig] = ratio_from * from[ig] + ratio_to * to[ig];
	time_vertex[ib] = ratio_from * from[ib] + ratio_to * to[ib];
	time_vertex[ia] = ratio_from * from[ia] + ratio_to * to[ia];
	
	//下の点の設定
	time_vertex[VCOUNT + ix] = ratio_from * from[VCOUNT + ix] + ratio_to * to[VCOUNT + ix];
	time_vertex[VCOUNT + iy] = ratio_from * from[VCOUNT + iy] + ratio_to * to[VCOUNT + iy];
	time_vertex[VCOUNT + iz] = ratio_from * from[VCOUNT + iz] + ratio_to * to[VCOUNT + iz];
	time_vertex[VCOUNT + it] = time;
	time_vertex[VCOUNT + ir] = ratio_from * from[VCOUNT + ir] + ratio_to * to[VCOUNT + ir];
	time_vertex[VCOUNT + ig] = ratio_from * from[VCOUNT + ig] + ratio_to * to[VCOUNT + ig];
	time_vertex[VCOUNT + ib] = ratio_from * from[VCOUNT + ib] + ratio_to * to[VCOUNT + ib];
	time_vertex[VCOUNT + ia] = ratio_from * from[VCOUNT + ia] + ratio_to * to[VCOUNT + ia];
	
	
}



__host__
__device__
void updateTimeIndex
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_index,
		unsigned int end_index,
		unsigned int* const time_index,
		float* const time_vertex
	)
{
	//インターリーブ配列としてのインデックスと
	//頂点配列としてのインデックスを区別するため、
	//接尾辞を下記にする
	//_index  : 頂点配列としてのインデックス
	//_iindex : インターリーブ配列としてのインデックス
	
	//区間に含まれているかをチェックするインデックス
	unsigned int v_iindex = VCOUNT * (*time_index);
	
	unsigned int begin_iindex = VCOUNT * begin_index;
	unsigned int   end_iindex = VCOUNT *   end_index;
	
	
	//最初の区間の手前の場合
	if(v_iindex < begin_iindex)
	{
		//最初の区間に含まれなければそのまま終了
		if(!isInInterval(time, vertex_d, begin_iindex, end_iindex, begin_iindex))
		{
			return;
		}
	}
	
	//最後の区間の先の場合はそのまま終了
	if(v_iindex >= end_iindex)
	{
		return;
	}
	
	
	//time_indexから順に区間に含まれているかチェックし、
	//含まれている区間の情報から頂点を設定する
	for(; v_iindex < end_iindex; v_iindex+=VCOUNT2)
	{
		if(isInInterval(time, vertex_d, begin_iindex, end_iindex, v_iindex))
		{
			*time_index = v_iindex / VCOUNT;
			updateTimeVertex(time, vertex_d, begin_iindex, end_iindex, v_iindex, time_vertex);
			return;
		}
	}
	//見つからなかった場合は最後のインデックスを設定する
	*time_index = end_iindex / VCOUNT;
}

//elementを更新する
__host__
__device__
void updateElement
	(
		const unsigned int* const index_list_d,
		unsigned int path_index,
		unsigned int* const element_d
	)
{
	unsigned int begin_index = index_list_d[3 * path_index + 0];
	unsigned int  past_index = index_list_d[3 * path_index + 1];
	unsigned int   now_index = index_list_d[3 * path_index + 2];
	
	unsigned int end_index = index_list_d[3 * (path_index + 1)];
	
	unsigned int past_vertex_index = begin_index;
	unsigned int  now_vertex_index = begin_index + 2;
	
	unsigned int ei = begin_index;
	element_d[ei] = past_vertex_index; ei++;
	element_d[ei] = past_vertex_index; ei++;
	
	element_d[ei] = past_vertex_index;     ei++;
	element_d[ei] = past_vertex_index + 1; ei++;
	
	unsigned int sei = past_index + 2;
	for(; (ei < end_index) && (sei < now_index + 2); ei++, sei++)
	{
		element_d[ei] = sei;
	}
	if(ei < end_index)
	{
		element_d[ei] = now_vertex_index;
		ei++;
	}
	for(;ei < end_index;ei++)
	{
		element_d[ei] = now_vertex_index + 1;
	}
}

/**
 * @brief nowに応じてVAOを更新する
 *
 * block 1つにpath 1つが対応する
 *
 * @param now
 * @param vertex_d
 * @param element_d
 * @param index_list_d
 * @param path_count
 *
 * @return 
 */
__global__
void updateDeviceDataCUDA
	(
		float now,
		float past,
		float* const vertex_d,
		unsigned int* const element_d,
		unsigned int* const index_list_d,
		unsigned int path_count
	)
{
	//pathのインデックス
	unsigned int pi = blockIdx.x;
	
	//path[pi]に関連するインデックス
	unsigned int begin_index = index_list_d[3 * pi    ];
	unsigned int* const past_index  = &index_list_d[3 * pi + 1];
	unsigned int* const now_index   = &index_list_d[3 * pi + 2];
	
	//最後の次のインデックス
	unsigned int end_index = index_list_d[3 * (pi + 1)];
	
	//path[pi]の頂点数
	//unsigned int vertex_count = end_index - begin_index;
	
	//pastの頂点
	float* const past_vertex = &vertex_d[begin_index];
	
	//nowの頂点
	float* const now_vertex = &vertex_d[begin_index + VCOUNT2];
	
	//pastIndexを更新する
	updateTimeIndex(past, vertex_d, begin_index + 4, end_index, past_index, past_vertex);
	
	//nowIndexを更新する
	updateTimeIndex( now, vertex_d, begin_index + 4, end_index,  now_index,  now_vertex);
	
	//elementを更新する
	updateElement(index_list_d, pi, element_d);
}


void PathsVAO::updateDeviceData(time_t now)
{
	vaoType& vao = doubleBufferingCurrent->vao;
	DeviceMemory<unsigned int>& indexListDevice = doubleBufferingCurrent->indexListDevice;
	
	//VAOをCUDA向けに確保
	Map<vaoType> m(doubleBufferingCurrent->vao);
	
	//VAOのデバイスメモリを取得
	float* v_d = vao.getVertexDevicePointer();
	unsigned int* e_d = vao.getElementDevicePointer();
	
	//indexListDeviceのデバイスメモリを取得
	unsigned int* il_d = indexListDevice.getDevicePointer();
	
	//パスの個数
	unsigned int path_count = ( indexListDevice.getCount() / 3 ) - 1;
	
	//ここにCUDA関数を書く予定(多分引数足りない)
	dim3 grid(path_count, 1, 1);
	dim3 block(1,1,1);
	updateDeviceDataCUDA<<<grid, block>>>
		(
			static_cast<float>(now),
			static_cast<float>(now - this->drawTimeWidth),
			v_d,
			e_d,
			il_d,
			path_count
		);
	
}



