#include "hip/hip_runtime.h"

#include "PathsVAO.h"


#define VCOUNT (9)
#define VCOUNT2 (18)




__device__
bool isInInterval
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_index,
		unsigned int end_index,
		unsigned int  vertex_index
	)
{
	const unsigned int it = 3;
	if((vertex_index < begin_index) || (end_index <= vertex_index))
	{
		return false;
	}
	if((vertex_d[vertex_index + it] <= time) && (time < vertex_d[vertex_index + VCOUNT2 + it]))
	{
		return true;
	}
	return false;
}

__device__
void updateTimeVertex
	(
	 float time,
	 const float* const vertex_d,
	 unsigned int begin_index,
	 unsigned int end_index,
	 unsigned int vi,
	 float* const time_vertex
	)
{
	//各座標を示すインデックス
	const unsigned int ix = 0;
	const unsigned int iy = 1;
	const unsigned int iz = 2;
	const unsigned int it = 3;
	const unsigned int ir = 4;
	const unsigned int ig = 5;
	const unsigned int ib = 6;
	const unsigned int ia = 7;
	
	const float* const from = &vertex_d[vi          ];
	const float* const to   = &vertex_d[vi + VCOUNT2];
	float from_time = from[it];
	float   to_time =   to[it];

	//内分比
	float ratio_from = 1.0 - ((float)(time - from_time))/((float)(to_time - from_time));
	float ratio_to = 1.0 - ratio_from;
	
	//上の点の設定
	time_vertex[ix] = ratio_from * from[ix] + ratio_to * to[ix];
	time_vertex[iy] = ratio_from * from[iy] + ratio_to * to[iy];
	time_vertex[iz] = ratio_from * from[iz] + ratio_to * to[iz];
	time_vertex[it] = time;
	time_vertex[ir] = ratio_from * from[ir] + ratio_to * to[ir];
	time_vertex[ig] = ratio_from * from[ig] + ratio_to * to[ig];
	time_vertex[ib] = ratio_from * from[ib] + ratio_to * to[ib];
	time_vertex[ia] = ratio_from * from[ia] + ratio_to * to[ia];
	
	//下の点の設定
	time_vertex[VCOUNT + ix] = ratio_from * from[VCOUNT + ix] + ratio_to * to[VCOUNT + ix];
	time_vertex[VCOUNT + iy] = ratio_from * from[VCOUNT + iy] + ratio_to * to[VCOUNT + iy];
	time_vertex[VCOUNT + iz] = ratio_from * from[VCOUNT + iz] + ratio_to * to[VCOUNT + iz];
	time_vertex[VCOUNT + it] = time;
	time_vertex[VCOUNT + ir] = ratio_from * from[VCOUNT + ir] + ratio_to * to[VCOUNT + ir];
	time_vertex[VCOUNT + ig] = ratio_from * from[VCOUNT + ig] + ratio_to * to[VCOUNT + ig];
	time_vertex[VCOUNT + ib] = ratio_from * from[VCOUNT + ib] + ratio_to * to[VCOUNT + ib];
	time_vertex[VCOUNT + ia] = ratio_from * from[VCOUNT + ia] + ratio_to * to[VCOUNT + ia];
	
	
}



__device__
void updateTimeIndex
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_index,
		unsigned int end_index,
		unsigned int* const time_index,
		float* const time_vertex
	)
{
	//区間に含まれているかをチェックするインデックス
	unsigned int vi = *time_index;
	
	//最初の区間の手前の場合
	if(vi < begin_index)
	{
		//最初の区間に含まれなければそのまま終了
		if(!isInInterval(time, vertex_d, begin_index, end_index, begin_index))
		{
			return;
		}
	}
	
	//最後の区間の先の場合はそのまま終了
	if(vi >= end_index)
	{
		return;
	}
	
	
	//time_indexから順に区間に含まれているかチェックし、
	//含まれている区間の情報から頂点を設定する
	for(; vi < end_index; vi+=VCOUNT2)
	{
		if(isInInterval(time, vertex_d, begin_index, end_index, vi))
		{
			*time_index = vi;
			updateTimeVertex(time, vertex_d, begin_index, end_index, vi, time_vertex);
			return;
		}
	}
	//見つからなかった場合は最後のインデックスを設定する
	*time_index = end_index;
}

//elementを更新する
__device__
void updateElement
	(
		const unsigned int* const index_list_d,
		unsigned int path_index,
		unsigned int* const element_d
	)
{
	unsigned int begin_index = index_list_d[3 * path_index + 0];
	unsigned int  past_index = index_list_d[3 * path_index + 1];
	unsigned int   now_index = index_list_d[3 * path_index + 2];
	
	unsigned int end_index = index_list_d[3 * (path_index + 1)];
	
	unsigned int past_vertex_index = begin_index;
	unsigned int  now_vertex_index = begin_index + 2;
	
	unsigned int ei = begin_index;
	element_d[ei] = past_vertex_index; ei++;
	element_d[ei] = past_vertex_index; ei++;
	
	element_d[ei] = past_vertex_index;     ei++;
	element_d[ei] = past_vertex_index + 1; ei++;
	
	unsigned int sei = past_index + 2;
	for(; (ei < end_index) && (sei < now_index + 2); ei++, sei++)
	{
		element_d[ei] = sei;
	}
	if(ei < end_index)
	{
		element_d[ei] = now_vertex_index;
		ei++;
	}
	for(;ei < end_index;ei++)
	{
		element_d[ei] = now_vertex_index + 1;
	}
}

/**
 * @brief nowに応じてVAOを更新する
 *
 * block 1つにpath 1つが対応する
 *
 * @param now
 * @param vertex_d
 * @param element_d
 * @param index_list_d
 * @param path_count
 *
 * @return 
 */
__global__
void updateDeviceDataCUDA
	(
		float now,
		float past,
		float* const vertex_d,
		unsigned int* const element_d,
		unsigned int* const index_list_d,
		unsigned int path_count
	)
{
	//pathのインデックス
	unsigned int pi = blockIdx.x;
	
	//path[pi]に関連するインデックス
	unsigned int begin_index = index_list_d[3 * pi    ];
	unsigned int* const past_index  = &index_list_d[3 * pi + 1];
	unsigned int* const now_index   = &index_list_d[3 * pi + 2];
	
	//最後の次のインデックス
	unsigned int end_index = index_list_d[3 * (pi + 1)];
	
	//path[pi]の頂点数
	//unsigned int vertex_count = end_index - begin_index;
	
	//pastの頂点
	float* const past_vertex = &vertex_d[begin_index];
	
	//nowの頂点
	float* const now_vertex = &vertex_d[begin_index + 9];
	
	//pastIndexを更新する
	updateTimeIndex(past, vertex_d, begin_index, end_index, past_index, past_vertex);
	
	//nowIndexを更新する
	updateTimeIndex( now, vertex_d, begin_index, end_index,  now_index,  now_vertex);
	
	//elementを更新する
	updateElement(index_list_d, pi, element_d);
}


void PathsVAO::updateDeviceData(time_t now)
{
	//VAOをCUDA向けに確保
	Map<vaoType> m(*vaoCurrent);
	
	//VAOのデバイスメモリを取得
	float* v_d = vaoCurrent->getVertexDevicePointer();
	unsigned int* e_d = vaoCurrent->getElementDevicePointer();
	
	//indexListDeviceのデバイスメモリを取得
	unsigned int* il_d = indexListDevice.getDevicePointer();
	
	//パスの個数
	unsigned int path_count = ( indexList.size() / 3 ) - 1;
	
	//ここにCUDA関数を書く予定(多分引数足りない)
	dim3 grid(path_count, 1, 1);
	dim3 block(1,1,1);
	updateDeviceDataCUDA<<<grid, block>>>
		(
			static_cast<float>(now),
			static_cast<float>(now - this->drawTimeWidth),
			v_d,
			e_d,
			il_d,
			path_count
		);
}



