
#include "PathsVAO.h"
#include "PathsVAO_cuda_kernel.cuh"


void PathsVAO::updateDeviceData(time_t now)
{
	vaoType& vao = doubleBufferingCurrent->vao;
	DeviceMemory<unsigned int>& indexListDevice = doubleBufferingCurrent->indexListDevice;
	
	//VAOをCUDA向けに確保
	Map<vaoType> m(vao);
	
	//VAOのデバイスメモリを取得
	float* v_d = vao.getVertexDevicePointer();
	unsigned int* e_d = vao.getElementDevicePointer();
	
	//indexListDeviceのデバイスメモリを取得
	unsigned int* il_d = indexListDevice.getDevicePointer();
	
	//パスの個数
	unsigned int path_count = ( indexListDevice.getCount() / 4 ) - 1;
	
	//ここにCUDA関数を書く予定
	dim3 grid(path_count, 1, 1);
	dim3 block(1,1,1);
	PathsVAO_updateDeviceDataCUDA<<<grid, block>>>
		(
			static_cast<float>(now),
			static_cast<float>(now - this->drawTimeWidth),
			v_d,
			e_d,
			il_d,
			path_count
		);
	//CUDA関数が完了するまで待機
	hipDeviceSynchronize();
}



