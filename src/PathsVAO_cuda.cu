#include "hip/hip_runtime.h"

#include "PathsVAO.h"


__global__
void updateDeviceDataCUDA(float* vertex_d, unsigned int* index_list_d, unsigned int path_count)
{
	
}


void PathsVAO::updateDeviceData(time_t now)
{
	//VAOをCUDA向けに確保
	Map<vaoType> m(*vaoCurrent);
	//VAOのデバイスメモリを取得
	float* v_d = vaoCurrent->getVertexDevicePointer();
	//indexListDeviceのデバイスメモリを取得
	unsigned int* il_d = indexListDevice.getDevicePointer();
	
	//パスの個数
	unsigned int path_count = ( indexList.size() / 3 ) - 1;
	
	//ここにCUDA関数を書く予定(多分引数足りない)
	updateDeviceDataCUDA<<<1,1>>>(v_d, il_d, path_count);
}



