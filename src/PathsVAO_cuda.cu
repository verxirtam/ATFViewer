
#include "PathsVAO.h"
#include "PathsVAO_cuda_kernel.cuh"


void PathsVAO::updateDeviceData(time_t now)
{
	
	////////////////////////////////////////
	// テスト
	////////////////////////////////////////
	//GLの描画が完了するまで待機
	glFinish();
	////////////////////////////////////////
	
	vaoType& vao = doubleBufferingCurrent->vao;
	DeviceMemory<unsigned int>& indexListDevice = doubleBufferingCurrent->indexListDevice;
	
	//VAOをCUDA向けに確保
	Map<vaoType> m(vao);
	
	//VAOのデバイスメモリを取得
	float* v_d = vao.getVertexDevicePointer();
	unsigned int* e_d = vao.getElementDevicePointer();
	
	//indexListDeviceのデバイスメモリを取得
	unsigned int* il_d = indexListDevice.getDevicePointer();
	
	//パスの個数
	unsigned int path_count = ( indexListDevice.getCount() / 4 ) - 1;
	
	//std::cout << "path_count = " << path_count << std::endl;
	
	
	////////////////////////////////////////
	// テスト
	////////////////////////////////////////
	static bool output = true;
	std::vector<unsigned int> il_h_before;
	std::vector<unsigned int> e_h_before;
	if(output)
	{
		output = false;
		//indexListの取得
		std::vector<unsigned int> il_h(indexListDevice.getCount(),0);
		indexListDevice.memcpyDeviceToHost(il_h.data());
		il_h_before = il_h;
		
		//elementの取得
		std::vector<unsigned int> e_h(vao.getElementCount(), 0);
		hipMemcpy(e_h.data(), e_d, e_h.size() * sizeof(float), hipMemcpyDeviceToHost);
		e_h_before = e_h;
	}
	
	////////////////////////////////////////
	
	TimeSeparation::Position position = TimeSeparation::Position::current;
	
	time_t time_start = 0;
	time_t time_end   = 0;
	
	//時間の区間の開始・終了時刻を取得する
	this->timeSeparation.getIntervalStatus(position, time_start, time_end);
	
	//DBから取得した時刻の範囲を取得する
	time_start -= this->drawTimeWidth;
	//time_end   += this->drawTimeWidth;
	
	float  now_float = static_cast<float>(now - time_start);
	float past_float = static_cast<float>(now - this->drawTimeWidth - time_start);
	
	//ここにCUDA関数を書く予定
	dim3 grid(path_count, 1, 1);
	dim3 block(1,1,1);
	PathsVAO_updateDeviceDataCUDA<<<grid, block>>>
		(
			now_float,
			past_float,
			v_d,
			e_d,
			il_d,
			path_count
		);
	//CUDA関数が完了するまで待機
	hipDeviceSynchronize();
	
	////////////////////////////////////////
	// テスト
	////////////////////////////////////////
	static bool output1 = true;
	if(output1)
	{
		output1 = false;
		//indexListの取得
		std::vector<unsigned int> il_h(indexListDevice.getCount(),0);
		indexListDevice.memcpyDeviceToHost(il_h.data());
		
		int imax = indexListDevice.getCount() / 4;
		std::cout << "////////////////////////////////////////" << std::endl;
		std::cout << "indexList: " << imax << std::endl;
		for(int i = 0; i < imax; i++)
		{
			bool equal = true;
			std::cout << i << ": ";
			for(int j = 0; j < 4; j++)
			{
				std::cout << il_h_before[i * 4 + j] << " ";
			}
			std::cout << " | ";
			for(int j = 0; j < 4; j++)
			{
				std::cout << il_h[i * 4 + j] << " ";
				
				if(il_h_before[i * 4 + j] != il_h[i * 4 + j])
				{
					equal = false;
				}
			}
			std::cout << " | ";
			if(!equal)
			{
				std::cout << " NOT EQUAL ";
			}
			std::cout << std::endl;
		}
		std::cout << "////////////////////////////////////////" << std::endl;
		
		//elementの取得
		std::vector<unsigned int> e_h(vao.getElementCount(), 0);
		hipMemcpy(e_h.data(), e_d, e_h.size() * sizeof(float), hipMemcpyDeviceToHost);
		int kmax = imax - 1;
		for(int k = 0; k < kmax; k++)
		{
			bool equal = true;
			
			std::cout << k << ": ";
			int lmin = il_h[ k      * 4 + 3];
			int lmax = il_h[(k + 1) * 4 + 3];
			for(int l = lmin; l < lmax; l++)
			{
				std::cout << e_h_before[l] << ", ";
			}
			std::cout << std::endl;
			
			std::cout << k << ": ";
			lmin = il_h[ k      * 4 + 3];
			lmax = il_h[(k + 1) * 4 + 3];
			for(int l = lmin; l < lmax; l++)
			{
				std::cout << e_h[l] << ", ";
				
				if(e_h_before[l] != e_h[l])
				{
					equal = false;
				}
			}
			std::cout << std::endl;
			std::cout << k << ": ";
			if(!equal)
			{
				std::cout << "NOT EQUAL";
			}
			std::cout << std::endl;
		}
	}
	
	////////////////////////////////////////
}



