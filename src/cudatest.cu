/*
 * =====================================================================================
 *
 *       Filename:  cudatest.cu
 *
 *    Description:  cudaの練習用
 *
 *        Version:  1.0
 *        Created:  2016年04月23日 23時47分33秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */

#include <iostream>
#include <ctime>

#include <hip/hip_runtime.h>

using namespace std;


#define MIN (0)
#define MAX (20)

__device__ void countCrossing(int* r, int j_start, int j_end)
{
	
	if(j_start != j_end)
	{
		int diff = (j_start < j_end) ? 1 : -1;
		int kmax = (j_start < j_end) ? (j_end - j_start) : (j_start - j_end);
		for(int k = 0; k < kmax; k++)
		{
			int j = j_start + (k + 1) * diff;
			int j_index = j * 2;
			j_index += diff > 0 ? 0 : 1;
			
			if((MIN <= j_index) && (j_index < MAX))
			{
				atomicAdd(&(r[j_index]),1);
			}
		}
	}
}

//ダミーのカーネル関数
__global__ void dummy(void)
{
}


//floor(w[i])の個数を集計する
//メモリアクセスの工夫はしない
__global__ void getIndex_01Simple(float* w, int* r)
{
	int i_start = (threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x ) * blockDim.x) * 3;
	int i_end   = i_start + 1;
	
	int j_start = (int)floorf(w[i_start]);
	int j_end   = (int)floorf(w[i_end  ]);
	
	countCrossing(r, j_start, j_end);
}

__global__ void getIndex_02Coalesing(float* w, int* r)
{
	//スレッドインデックス、ブロックインデックスの読み替え
	//（可読性のため）
	int p = threadIdx.x;
	int x = blockIdx.x;
	int y = blockIdx.y;
	int L = blockDim.x;
	int M = gridDim.x;
	
	
	//シェアードメモリ
	//グローバル変数のアクセス用
	extern __shared__ float w_s[];
	
	//グローバルメモリからシェアードメモリにコピー
	//コアレスアクセスになるようにスレッドごとに隣接するアドレスを指定する
	//スレッド数の3倍の個数があるので3回に分ける
	int offset = 3 * (x + y * M) * L;
	w_s[p        ] = w[offset + p        ];
	w_s[p +     L] = w[offset + p +     L];
	w_s[p + 2 * L] = w[offset + p + 2 * L];
	
	//スレッドが参照するシェアードメモリのインデックス
	//32bit*3おきなのでバンクコンフリクトは起きない
	int i_s_start = 3 * p;
	int i_s_end   = 3 * p + 1;
	
	//シェアードメモリの参照
	int j_start = (int)floorf(w_s[i_s_start]);
	int j_end   = (int)floorf(w_s[i_s_end  ]);
	
	countCrossing(r, j_start, j_end);
}

template <typename T>
class HostDeviceSeq
{
private:
	int count;
	T* host;
	T* device;
public:
	HostDeviceSeq(int c):host(NULL),device(NULL)
	{
		count = c;
		host = new T[count];
		hipMalloc((void**)&device, count * sizeof(T));
	}
	~HostDeviceSeq()
	{
		hipFree(device);
		delete [] host;
	}
	T& operator[](int i)
	{
		return host[i];
	}
	void memcpyHostToDevice(void)
	{
		hipMemcpy(device, host, count * sizeof(T), hipMemcpyHostToDevice);
	}
	void memcpyDeviceToHost(void)
	{
		hipMemcpy(host, device, count * sizeof(T), hipMemcpyDeviceToHost);
	}
	T* getDeviceAddress()
	{
		return device;
	}
};

int main(int argc, char const* argv[])
{
	dim3 blocks(2,2,1);
	dim3 threads(33,1,1);
	//dim3 blocks(512,N/512/512,1);
	//dim3 threads(512,1,1);
	
	const int N = threads.x * blocks.x * blocks.y;
	//const int N = 512 * 512 * 32;
	const int RSIZE = MAX - MIN;
	
	
	HostDeviceSeq<float> w(3 * N);
	HostDeviceSeq<int> r01(RSIZE);
	HostDeviceSeq<int> r02(RSIZE);
	
	
	
	srand((unsigned int)time(NULL));
	
	for(int i = 0; i < N; i++)
	{
		w[3 * i    ] = ((float)rand()) / ((float)(RAND_MAX) + 1.0f) * 10.0f;
		w[3 * i + 1] = ((float)rand()) / ((float)(RAND_MAX) + 1.0f) * 10.0f;
		w[3 * i + 2] = 0.0f;//パディング用
	}
	
	for(int i = 0;i < RSIZE; i++)
	{
		r01[i] = 0;
		r02[i] = 0;
	}

	
	
	//ダミーのカーネル関数
	dummy<<< blocks, threads >>>();
	
	clock_t start01 = clock();
	w.memcpyHostToDevice();
	r01.memcpyHostToDevice();
	getIndex_01Simple<<< blocks, threads >>>(w.getDeviceAddress(), r01.getDeviceAddress());
	r01.memcpyDeviceToHost();
	clock_t end01 = clock();
	
	clock_t start02 = clock();
	w.memcpyHostToDevice();
	r02.memcpyHostToDevice();
	int size = 3 * threads.x * sizeof(float);
	getIndex_02Coalesing<<< blocks, threads, size >>>(w.getDeviceAddress(), r02.getDeviceAddress());
	r02.memcpyDeviceToHost();
	clock_t end02 = clock();
	
	cout << "N = " << N << endl;
	if(false)
	{
		cout << "w:\t";
		for(int i = 0;i < 3 * N; i++)
		{
			cout << w[i] << "\t";
		}
		cout << endl;
	}
	cout << "r:\t";
	for(int i = 0;i < RSIZE; i++)
	{
		cout << r01[i] << "\t";
	}
	cout << endl;
	
	cout << "getIndex_01Simple()経過時間 = " << (double)(end01 - start01) / CLOCKS_PER_SEC << "sec." << endl;
	cout << "getIndex_02Coalesing()経過時間 = " << (double)(end02 - start02) / CLOCKS_PER_SEC << "sec." << endl;
	
	bool result = true;
	for(int i = 0;i < RSIZE; i++)
	{
		if(r01[i]!=r02[i])
		{
			result = false;
		}
	}
	if(result)
	{
		cout << "getIndex_02Coalesing()の実行に成功しました" << endl;
	}
	else
	{
		cout << "getIndex_02Coalesing()の結果が不正です" << endl;
	}
	
	return 0;
}

