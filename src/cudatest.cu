#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  cudatest.cu
 *
 *    Description:  cudaの練習用
 *
 *        Version:  1.0
 *        Created:  2016年04月23日 23時47分33秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */

#include <iostream>
#include <ctime>
#include <cmath>

#include <hip/hip_runtime.h>

using namespace std;


#define MIN (0)
#define MAX (20)
#define RSIZE (20)

__device__ void countCrossing(int* r, int j_start, int j_end)
{
	
	if(j_start != j_end)
	{
		int diff = (j_start < j_end) ? 1 : -1;
		int kmax = (j_start < j_end) ? (j_end - j_start) : (j_start - j_end);
		for(int k = 0; k < kmax; k++)
		{
			int j = j_start + (k + 1) * diff;
			int j_index = j * 2;
			j_index += diff > 0 ? 0 : 1;
			
			if((MIN <= j_index) && (j_index < MAX))
			{
				atomicAdd(&(r[j_index]),1);
			}
		}
	}
}

//ダミーのカーネル関数
__global__ void dummy(void)
{
}


//比較用の何もしないカーネル関数
//データ転送のみ実施
__global__ void getIndex_00DoNothing(float* w, int* r)
{
	//何もしない
}
//floor(w[i])の個数を集計する
//メモリアクセスの工夫はしない
__global__ void getIndex_01Simple(float* w, int* r)
{
	int i_start = (threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x ) * blockDim.x) * 3;
	int i_end   = i_start + 1;
	
	int j_start = (int)floorf(w[i_start]);
	int j_end   = (int)floorf(w[i_end  ]);
	
	countCrossing(r, j_start, j_end);
}

__device__ void addResult(int* r_t, int s, int t)
{
	int s2 = (s >> 1);
	if(t < RSIZE * s2)
	{
		r_t[t] += r_t[t + RSIZE * s2];
	}
}

__global__ void getIndex_02Coalesing(float* w, int* r_t, int _2_B, int* r)
{
	//スレッドインデックス、ブロックインデックスの読み替え
	//（可読性のため）
	int p = threadIdx.x;//ブロック内のスレッドのインデックス
	int x = blockIdx.x;//ブロックのインデックスのx成分
	int y = blockIdx.y;//ブロックのインデックスのy成分
	int L = blockDim.x;//ブロック1つあたりのx方向のスレッド数
	int M = gridDim.x;//x方向のブロック数
	
	int b = x + y * M;//ブロックの通し番号
	int t = p + b * L;//スレッドの通し番号
	
	//シェアードメモリ
	//グローバル変数のアクセス用
	extern __shared__ float w_s[];
	
	//グローバルメモリからシェアードメモリにコピー
	//コアレスアクセスになるようにスレッドごとに隣接するアドレスを指定する
	//スレッド数の3倍の個数があるので3回に分ける
	int offset = 3 * (x + y * M) * L;
	w_s[p        ] = w[offset + p        ];
	w_s[p +     L] = w[offset + p +     L];
	w_s[p + 2 * L] = w[offset + p + 2 * L];
	
	//ブロック内のスレッドについて同期する
	//上記のシェアードメモリへのコピーが
	//ブロック内の全てのスレッドで完了するまで待機する
	//ブロック内で起動するスレッドがワープのサイズ（GTX260なら32）
	//を超えるとスレッドが同期して実行されることが保証されなくなる
	//逆に言うと同じワープ内のスレッドは常に同期されているので、
	//工夫すれば__syncthreads()を使用した明示的な同期を不要に出来る可能性がある
	__syncthreads();
	
	//スレッドが参照するシェアードメモリのインデックス
	//32bit*3おきなのでバンクコンフリクトは起きない
	int i_s_start = 3 * p;
	int i_s_end   = 3 * p + 1;
	
	//シェアードメモリの参照
	int j_start = (int)floorf(w_s[i_s_start]);
	int j_end   = (int)floorf(w_s[i_s_end  ]);
	
	
	//shared memory r_s alloc
	__shared__ int r_s[RSIZE];
	
	//copy global memory r to shared memory r_s
	//RSIZE <= L であることを前提にする
	//一部だけコピー
	if(p < RSIZE)
	{
		r_s[p] = r[p];
	}
	//シェアードメモリのコピー中に
	//別のワープのスレッドが集計を始めないように同期する
	__syncthreads();

	//シェアードメモリr_sを使用して集計
	//集計時にブロック内でしか衝突は起きない
	countCrossing(r_s, j_start, j_end);
	
	//集計が終わる前にグローバルメモリr_tへのコピーが
	//始まらないように同期する
	__syncthreads();
	
	//copy shared memory r_s to global memory r
	//シェアードメモリから集計用のグローバルメモリr_tへコピー
	if(p < RSIZE)
	{
		r_t[p + b * RSIZE] = r_s[p];
	}
	//グローバルメモリr_tへのコピーが終わる前に
	//集計が始まらないようにブロック間で同期する
	//そのために一旦カーネルを終了する
	
	
	return;
}


//r_t	集計する値の格納された配列
//    	大きさRSIZEのベクトルが_2_B個格納されている
//_2_B	ベクトルの個数(COUNTPERBLOCKで割り切れること)
//r   	集計結果を格納するベクトル（大きさRSIZE * _2_B / COUNTPERBLOCK）
template <typename COUNTPERBLOCK>//ブロックあたりのベクトルの個数
__global__ void getIndex_02Coalesing_02Accumulate(int* r_t, r_t_copy, int _2_B, int* r)
{
	//スレッドインデックス、ブロックインデックスの読み替え
	//（可読性のため）
	int p = threadIdx.x;//ブロック内のスレッドのインデックス
	int x = blockIdx.x;//ブロックのインデックスのx成分
	int y = blockIdx.y;//ブロックのインデックスのy成分
	int L = blockDim.x;//ブロック1つあたりのx方向のスレッド数
	int M = gridDim.x;//x方向のブロック数
	
	int b = x + y * M;//ブロックの通し番号
	int t = p + b * L;//スレッドの通し番号
	
	
	
	int block_size = COUNTPERBLOCK * RSIZE;
	//所属するブロックのシェアードメモリ
	int r_t_b[COUNTPERBLOCK * RSIZE];

	
	//生成するブロック数
	int block_count = _2_B / COUNTPERBLOCK;
	
	//当該ブロックがアクセスする開始インデックス
	int start_index = b * block_size;

	//結果の格納先の開始インデックス
	int result_start_index = b * RSIZE;
	
	//ブロック内での集計を行う
	//r_t + start_index: 開始インデックス
	//r + result_start_index: 結果の格納先の開始アドレス
	//COUNTPERBLOCK: ブロック内のベクトルの個数
	//r_t_b: ブロック内で使用するシェアードメモリ（サイズ：COUNTPERBLOCK * RSIZE）
	getIndex_02Coalesing_02Accumulate_01ReduceAtBlock(r_t + start_index, r + result_start_index, COUNTPERBLOCK, r_t_b);
	
	//ブロック間の同期がいるためここでカーネルを終了する
	
	//このカーネルが終了した時点で各ブロックの集計が完了しr[]に格納されている
	//結果はblock_count個のベクトルなので、この関数をblock_count==1になるまで繰り返して集計結果を得る
}

template <typename T>
class HostDeviceSeq
{
private:
	int count;
	T* host;
	T* device;
public:
	HostDeviceSeq(int c):host(NULL),device(NULL)
	{
		count = c;
		host = new T[count];
		hipMalloc((void**)&device, count * sizeof(T));
	}
	~HostDeviceSeq()
	{
		hipFree(device);
		delete [] host;
	}
	T& operator[](int i)
	{
		return host[i];
	}
	void memcpyHostToDevice(void)
	{
		hipMemcpy(device, host, count * sizeof(T), hipMemcpyHostToDevice);
	}
	void memcpyDeviceToHost(void)
	{
		hipMemcpy(host, device, count * sizeof(T), hipMemcpyDeviceToHost);
	}
	T* getDeviceAddress()
	{
		return device;
	}
};

int main(int argc, char const* argv[])
{
	dim3 blocks(8,2,1);
	//dim3 blocks(128,64,1);
	dim3 threads(512,1,1);
	//dim3 blocks(512,N/512/512,1);
	//dim3 threads(512,1,1);
	
	const int N = threads.x * blocks.x * blocks.y;
	//const int N = 512 * 512 * 32;
	//const int RSIZE = MAX - MIN;
	
	
	HostDeviceSeq<float> w(3 * N);
	HostDeviceSeq<int> r00(RSIZE);
	HostDeviceSeq<int> r01(RSIZE);
	HostDeviceSeq<int> r02(RSIZE);
	
	int B = (int)ceil(log(blocks.x * blocks.y)/log(2.0f));
	int _2_B = (1 << B);//2^B
	int r_t_size = RSIZE * (_2_B);//RSIZE * 2^B
	HostDeviceSeq<int> r_t(r_t_size);
	
	
	srand((unsigned int)time(NULL));
	
	for(int i = 0; i < N; i++)
	{
		w[3 * i    ] = ((float)rand()) / ((float)(RAND_MAX) + 1.0f) * 10.0f;
		w[3 * i + 1] = ((float)rand()) / ((float)(RAND_MAX) + 1.0f) * 10.0f;
		w[3 * i + 2] = 0.0f;//パディング用
	}
	
	for(int i = 0;i < RSIZE; i++)
	{
		r00[i] = 0;
		r01[i] = 0;
		r02[i] = 0;
	}

	for(int i = 0; i < r_t_size; i++)
	{
		r_t[i]=0;
	}
	
	
	
	
	
	
	//ダミーのカーネル関数
	dummy<<< 1, 1 >>>();
	
	//デバイスへのデータのコピー
	clock_t start_cpy = clock();
	w.memcpyHostToDevice();
	clock_t end_cpy = clock();
	
	clock_t start00 = clock();
	r01.memcpyHostToDevice();
	getIndex_00DoNothing<<< blocks, threads >>>(w.getDeviceAddress(), r00.getDeviceAddress());
	r01.memcpyDeviceToHost();
	clock_t end00 = clock();
	
	clock_t start01 = clock();
	r01.memcpyHostToDevice();
	getIndex_01Simple<<< blocks, threads >>>(w.getDeviceAddress(), r01.getDeviceAddress());
	r01.memcpyDeviceToHost();
	clock_t end01 = clock();
	
	clock_t start02 = clock();
	r02.memcpyHostToDevice();
	r_t.memcpyHostToDevice();
	int size = 3 * threads.x * sizeof(float);
	getIndex_02Coalesing<<< blocks, threads, size >>>(w.getDeviceAddress(), r_t.getDeviceAddress(), _2_B, r02.getDeviceAddress());
	r02.memcpyDeviceToHost();
	clock_t end02 = clock();
	
	
	
	cout << "N = " << N << endl;
	if(false)
	{
		cout << "w:\t";
		for(int i = 0;i < 3 * N; i++)
		{
			cout << w[i] << "\t";
		}
		cout << endl;
	}
	cout << "r01:\t";
	for(int i = 0;i < RSIZE; i++)
	{
		cout << r01[i] << "\t";
	}
	cout << endl;
	cout << "r02:\t";
	for(int i = 0;i < RSIZE; i++)
	{
		cout << r02[i] << "\t";
	}
	cout << endl;
	
	cout << "w.memcpyHostToDevice()経過時間 = " << (double)(end_cpy - start_cpy) / CLOCKS_PER_SEC << "sec." << endl;
	cout << "getIndex_00DoNothing()経過時間 = " << (double)(end00 - start00) / CLOCKS_PER_SEC << "sec." << endl;
	cout << "getIndex_01Simple()経過時間    = " << (double)(end01 - start01) / CLOCKS_PER_SEC << "sec." << endl;
	cout << "getIndex_02Coalesing()経過時間 = " << (double)(end02 - start02) / CLOCKS_PER_SEC << "sec." << endl;
	
	bool result = true;
	for(int i = 0;i < RSIZE; i++)
	{
		if(r01[i]!=r02[i])
		{
			result = false;
		}
	}
	if(result)
	{
		cout << "getIndex_02Coalesing()の実行に成功しました" << endl;
	}
	else
	{
		cout << "getIndex_02Coalesing()の結果が不正です" << endl;
	}
	
	//テンポラリ領域の内容の確認
	if(true)
	{
		r_t.memcpyDeviceToHost();
		cout << "r_t_size : " << r_t_size << endl;
		cout << "r_t[] : " << endl;
		for(int i = 0; i < _2_B; i++)
		{
			cout << "\t";
			for(int j = 0; j < RSIZE; j++)
			{
				int index = j + i * RSIZE;
				cout << r_t[index] << "\t";
			}
			cout << endl;
		}
		cout << endl;
	}
	return 0;
}

