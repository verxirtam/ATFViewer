#include "hip/hip_runtime.h"

#include "PathsVAO_cuda_kernel.cuh"


#define VCOUNT (9)
#define VCOUNT2 (18)

#define INDEXCOUNT (4)
#define INDEX_BEGIN (0)
#define INDEX_PAST  (1)
#define INDEX_NOW   (2)
#define INDEX_ELEMENTBEGIN (3)


//インターリーブ配列としてのインデックスと
//頂点配列としてのインデックスを区別するため、
//接尾辞を下記にする
//_index  : 頂点配列としてのインデックス
//_iindex : インターリーブ配列としてのインデックス


__host__
__device__
bool PathsVAO_isInInterval
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_iindex,
		unsigned int end_iindex,
		unsigned int  vertex_iindex
	)
{
	const unsigned int it = 3;
	if((vertex_iindex < begin_iindex) || (end_iindex <= vertex_iindex))
	{
		return false;
	}
	if((vertex_d[vertex_iindex + it] <= time) && (time < vertex_d[vertex_iindex + VCOUNT2 + it]))
	{
		return true;
	}
	return false;
}

__host__
__device__
void PathsVAO_updateTimeVertex
	(
	 float time,
	 const float* const vertex_d,
	 unsigned int begin_iindex,
	 unsigned int end_iindex,
	 unsigned int v_iindex,
	 float* const time_vertex
	)
{
	//各座標を示すインデックス
	const unsigned int ix = 0;
	const unsigned int iy = 1;
	const unsigned int iz = 2;
	const unsigned int it = 3;
	const unsigned int ir = 4;
	const unsigned int ig = 5;
	const unsigned int ib = 6;
	const unsigned int ia = 7;
	
	unsigned int from_iindex = (v_iindex < begin_iindex) ? begin_iindex : v_iindex;
	unsigned int   to_iindex = (v_iindex < begin_iindex) ? begin_iindex : (v_iindex + VCOUNT2);
	from_iindex = (end_iindex <= from_iindex) ? (end_iindex - VCOUNT2) : from_iindex;
	  to_iindex = (end_iindex <=   to_iindex) ? (end_iindex - VCOUNT2) :   to_iindex;
	
	const float* const from = &vertex_d[from_iindex];
	const float* const to   = &vertex_d[  to_iindex];
	float from_time = from[it];
	float   to_time =   to[it];

	//内分比
	float ratio_from = 1.0f - ((float)(time - from_time))/((float)(to_time - from_time));
	float ratio_to = 1.0f - ratio_from;
	
	//上の点の設定
	time_vertex[ix] = ratio_from * from[ix] + ratio_to * to[ix];
	time_vertex[iy] = ratio_from * from[iy] + ratio_to * to[iy];
	time_vertex[iz] = ratio_from * from[iz] + ratio_to * to[iz];
	time_vertex[it] = time;
	time_vertex[ir] = ratio_from * from[ir] + ratio_to * to[ir];
	time_vertex[ig] = ratio_from * from[ig] + ratio_to * to[ig];
	time_vertex[ib] = ratio_from * from[ib] + ratio_to * to[ib];
	time_vertex[ia] = ratio_from * from[ia] + ratio_to * to[ia];
	
	//下の点の設定
	time_vertex[VCOUNT + ix] = ratio_from * from[VCOUNT + ix] + ratio_to * to[VCOUNT + ix];
	time_vertex[VCOUNT + iy] = ratio_from * from[VCOUNT + iy] + ratio_to * to[VCOUNT + iy];
	time_vertex[VCOUNT + iz] = ratio_from * from[VCOUNT + iz] + ratio_to * to[VCOUNT + iz];
	time_vertex[VCOUNT + it] = time;
	time_vertex[VCOUNT + ir] = ratio_from * from[VCOUNT + ir] + ratio_to * to[VCOUNT + ir];
	time_vertex[VCOUNT + ig] = ratio_from * from[VCOUNT + ig] + ratio_to * to[VCOUNT + ig];
	time_vertex[VCOUNT + ib] = ratio_from * from[VCOUNT + ib] + ratio_to * to[VCOUNT + ib];
	time_vertex[VCOUNT + ia] = ratio_from * from[VCOUNT + ia] + ratio_to * to[VCOUNT + ia];
	
	
}



__host__
__device__
void PathsVAO_updateTimeIndex
	(
		float time,
		const float* const vertex_d,
		unsigned int begin_index,
		unsigned int end_index,
		unsigned int* const time_index,
		float* const time_vertex
	)
{
	//インターリーブ配列としてのインデックスと
	//頂点配列としてのインデックスを区別するため、
	//接尾辞を下記にする
	//_index  : 頂点配列としてのインデックス
	//_iindex : インターリーブ配列としてのインデックス
	
	//区間に含まれているかをチェックするインデックス
	unsigned int v_iindex = VCOUNT * (*time_index);
	
	unsigned int begin_iindex = VCOUNT * begin_index;
	unsigned int   end_iindex = VCOUNT *   end_index;
	
	
	//最初の区間の手前の場合
	if(v_iindex < begin_iindex)
	{
		//最初の区間の時刻より手前であれば最初の区間の1つ手前のインデックスを返す
		const unsigned int it = 3;
		//最初の区間の時刻
		float time_first = vertex_d[begin_iindex + it];
		if(time < time_first)
		{
			*time_index = (begin_iindex - VCOUNT2) / VCOUNT;
			return;
		}
	}
	
	//最後の区間の先の場合はそのまま終了
	if(v_iindex >= end_iindex)
	{
		return;
	}
	
	
	//time_indexから順に区間に含まれているかチェックし、
	//含まれている区間の情報から頂点を設定する
	for(; v_iindex < end_iindex; v_iindex+=VCOUNT2)
	{
		if(PathsVAO_isInInterval(time, vertex_d, begin_iindex, end_iindex, v_iindex))
		{
			*time_index = v_iindex / VCOUNT;
			PathsVAO_updateTimeVertex(time, vertex_d, begin_iindex, end_iindex, v_iindex, time_vertex);
			return;
		}
	}
	//見つからなかった場合は最後のインデックスを設定する
	*time_index = end_iindex / VCOUNT;
}

//elementを更新する
__host__
__device__
void PathsVAO_updateElement
	(
		const unsigned int* const index_list_d,
		unsigned int path_index,
		unsigned int* const element_d
	)
{
	unsigned int begin_index = index_list_d[INDEXCOUNT * path_index + INDEX_BEGIN       ];
	unsigned int  past_index = index_list_d[INDEXCOUNT * path_index + INDEX_PAST        ];
	unsigned int   now_index = index_list_d[INDEXCOUNT * path_index + INDEX_NOW         ];
	unsigned int element_begin_index = index_list_d[INDEXCOUNT * path_index + INDEX_ELEMENTBEGIN];
	
	unsigned int begin_index_main = begin_index + 4;
	
	unsigned int end_index         = index_list_d[INDEXCOUNT * (path_index + 1) + INDEX_BEGIN];
	unsigned int element_end_index = index_list_d[INDEXCOUNT * (path_index + 1) + INDEX_ELEMENTBEGIN];
	
	unsigned int past_vertex_index = begin_index;
	unsigned int  now_vertex_index = begin_index + 2;
	
	
	//pastとnowの位置関係を示す真偽値
	//pastは頂点配列の手前にある
	bool past_before = (past_index <  begin_index_main);
	//pastは頂点配列の後にある
	bool past_after  = (end_index  <= past_index );
	//nowは頂点配列の手前にある
	bool now_before  = (now_index  <  begin_index_main);
	//nowは頂点配列の後にある
	bool now_after   = (end_index  <= now_index  );
	
	//pastを描画するかどうか
	bool past_draw = !(past_before || past_after);
	//nowを描画するかどうか
	bool now_draw  = !(now_before  || now_after);
	
	//pastとnowの間の頂点配列の開始・終了のインデックス
	unsigned int main_begin_vertex_index = past_before ? begin_index_main : (past_after ? end_index : (past_index + 2));
	unsigned int   main_end_vertex_index = now_before  ? begin_index_main : (now_after  ? end_index : (now_index  + 2));
	
	//インデックス配列のカウンタ
	unsigned int ei = element_begin_index;
	
	//(最初に2つ設定するダミーのインデックス)
	// =(最初に描画する点のインデックス)を決定する
	unsigned int first_vertex_index = past_draw ? past_vertex_index : main_begin_vertex_index;
	
	//ダミーのインデックスを設定する
	element_d[ei] = first_vertex_index; ei++;
	element_d[ei] = first_vertex_index; ei++;
	//描画する頂点のインデックスを設定する
	if(past_draw)
	{
		element_d[ei] = past_vertex_index    ; ei++;
		element_d[ei] = past_vertex_index + 1; ei++;
	}
	unsigned int sei = main_begin_vertex_index;
	for(;(ei < element_end_index) && (sei < main_end_vertex_index); ei++, sei++)
	{
		element_d[ei] = sei;
	}
	if(now_draw)
	{
		element_d[ei] = now_vertex_index    ; ei++;
		element_d[ei] = now_vertex_index + 1; ei++;
	}
	//最後に描画した頂点のインデックスをダミーとして最後まで設定する
	unsigned int last_vertex_index = element_d[ei -1];
	for(;ei < element_end_index; ei++)
	{
		element_d[ei] = last_vertex_index;
	}


	/*
	element_d[ei] = past_vertex_index; ei++;
	element_d[ei] = past_vertex_index; ei++;
	
	element_d[ei] = past_vertex_index;     ei++;
	element_d[ei] = past_vertex_index + 1; ei++;
	
	unsigned int sei = past_index + 2;
	for(; (ei < element_end_index) && (sei < now_index + 2); ei++, sei++)
	{		element_d[ei] = sei;
	}
	if(ei < end_index)
	{
		element_d[ei] = now_vertex_index;
		ei++;
	}
	for(;ei < element_end_index;ei++)
	{
		element_d[ei] = now_vertex_index + 1;
	}
	*/
}

/**
 * @brief nowに応じてVAOを更新する
 *
 * block 1つにpath 1つが対応する
 *
 * @param now
 * @param vertex_d
 * @param element_d
 * @param index_list_d
 * @param path_count
 *
 * @return 
 */
__host__
__device__
void PathsVAO_updateDeviceDataCUDAMain
	(
		float now,
		float past,
		float* const vertex_d,
		unsigned int* const element_d,
		unsigned int* const index_list_d,
		unsigned int path_index
	)
{
	//path[path_index]に関連するインデックス
	unsigned int begin_index        =  index_list_d[INDEXCOUNT * path_index + INDEX_BEGIN];
	unsigned int* const past_index  = &index_list_d[INDEXCOUNT * path_index + INDEX_PAST ];
	unsigned int* const now_index   = &index_list_d[INDEXCOUNT * path_index + INDEX_NOW  ];
	
	//最後の次のインデックス
	unsigned int end_index = index_list_d[INDEXCOUNT * (path_index + 1) + INDEX_BEGIN];
	
	//path[path_index]の頂点数
	//unsigned int vertex_count = end_index - begin_index;
	
	//pastの頂点
	float* const past_vertex = &vertex_d[VCOUNT * begin_index];
	
	//nowの頂点
	float* const now_vertex = &vertex_d[VCOUNT * begin_index + VCOUNT2];
	
	//pastIndexを更新する
	PathsVAO_updateTimeIndex(past, vertex_d, begin_index + 4, end_index, past_index, past_vertex);
	
	//nowIndexを更新する
	PathsVAO_updateTimeIndex( now, vertex_d, begin_index + 4, end_index,  now_index,  now_vertex);
	
	//elementを更新する
	PathsVAO_updateElement(index_list_d, path_index, element_d);
}


__global__
void PathsVAO_updateDeviceDataCUDA
	(
		float now,
		float past,
		float* const vertex_d,
		unsigned int* const element_d,
		unsigned int* const index_list_d
	)
{
	//pathのインデックス
	unsigned int path_index = blockIdx.x;
	//メインの処理に渡す
	PathsVAO_updateDeviceDataCUDAMain(now, past, vertex_d, element_d, index_list_d, path_index);
}

