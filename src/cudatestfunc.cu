/*
 * =====================================================================================
 *
 *       Filename:  cudatestfunc.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016年05月17日 00時26分44秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */


#include <hip/hip_runtime.h>





__global__ void test0(float* d_a)
{
	int i = threadIdx.x;
	d_a[i] *= 2.0f;
}



void cudatestfunc(float* h_a, int n)
{
	float* d_a;
	hipMalloc((void**)&d_a, n * sizeof(float));
	hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
	test0<<<1,n>>>(d_a);
	hipMemcpy(h_a, d_a, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_a);
}
/*
__host__ __device__ int getIndex(float x, float interval)
{
	return (int)floorf(x / interval);
}

__host__ __device__ int* getIndex(const float* const x, const float* const interval, int* const xi)
{
	xi[0] = getIndex(x[0], interval[0]);
	xi[1] = getIndex(x[1], interval[1]);
	xi[2] = getIndex(x[2], interval[2]);
	xi[3] = getIndex(x[3], interval[3]);

	return xi;
}


__host__ __device__ int sign(int x)
{
	return (x > 0) ? 1 : ( (x < 0) ? -1 : 0 );
}

__host__ __device__ int* getDirection(const int* const from, const int* const to, int* const direction)
{
	direction[0] = sign(to[0] - from[0]);
	direction[1] = sign(to[1] - from[1]);
	direction[2] = sign(to[2] - from[2]);
	direction[3] = sign(to[3] - from[3]);
	return direction;
}


__host__ __device__ bool equals(const int* const x0, const int* const x1)
{
	bool ret = true;
	ret = (x0[0] == x1[0]) ? ret : false;
	ret = (x0[1] == x1[1]) ? ret : false;
	ret = (x0[2] == x1[2]) ? ret : false;
	ret = (x0[3] == x1[3]) ? ret : false;
	return ret;
}

//交点を算出する
__host__ __device__ float* getCrossPoint
	(
		const float* const from,//線分の始点
		const float* const to,//線分の終点
		const int* const p_i,//セルの番号
		const float* const interval,//セルの幅
		const int* const dir_i,//線分の方向
		int i,//交点を求める方向
		float* const cross//算出する交点
	)
{
	//交点を求める平面の座標
	float x = interval[i] * (p_i[i] + ((dir_i[i] + 1) / 2));
	//交点の線分のパラメータ
	float s = (x - from[i]) / (to[i] - from[i]);
	//交点の算出
	cross[0] = (to[0] - from[0]) * s + from[0];
	cross[1] = (to[1] - from[1]) * s + from[1];
	cross[2] = (to[2] - from[2]) * s + from[2];
	cross[3] = (to[3] - from[3]) * s + from[3];
	//i成分については誤差が出ないようにもともと求めていたxを使用する
	cross[i] = x;
	//交点を返却
	return cross;
}

//交点が指定したセルの指定した方向の境界にいるかどうか判定する
__host__ __device__ bool atBound
	(
		const float* const cross,//交点
		const int* const p_i,//セル番号
		const float* const interval,//セルの幅
		int i//方向
	)
{
	bool ret = true;
	for(int j = 0; j < 4;j++)
	{
		if(j != i)
		{
			if((cross[i] < interval[j] * p_i[j]) || (interval[j] * (p_i[j] + 1) <= cross[i]))
			{
				ret = false;
			}
		}
		else
		{
			if((cross[i]  < interval[j] * (p_i[j] - 1)) || (interval[j] * (p_i[j] + 2) <= cross[i]))
			{
				ret = false;
			}
		}
	}
	return ret;
}


//線分が通過するセル境界に対し回数をカウントアップする
__host__ __device__ void countCrossing
	(
		const float* const from,//線分の始点
		const float* const to,//線分の終点
		int* const c,//カウンタ
		int n,//カウンタの個数
		const float* const interval//セルの幅
	)
{

	//fromの属するセル番号を取得
	int from_i[4];
	getIndex(from, interval, from_i);
	//toの属するセル番号を取得
	int to_i[4];
	getIndex(to, interval, to_i);
	//線分の各方向の向き
	int direction_i[4];
	getDirection(from_i, to_i, direction_i);
	
	//どのセルにも進まない線分かチェック
	if((direction_i[0] == 0) && (direction_i[1] == 0) && (direction_i[2] == 0) && (direction_i[3] == 0))
	{
		//同じセルにとどまっている線分なので集計なしで終了
		return;
	}

	//ループ用のセル番号
	int p_i[4] = {from_i[0], from_i[1], from_i[2], from_i[3]};
	
	
	//fromからtoまでに通過するセルを求める
	do
	{
		//次に通過するセル
		int next_i[4] = {p_i[0], p_i[1], p_i[2], p_i[3]};
		//p_iの次に通過するセルnext_iを求める
		for(int i = 0; i < 4; i++)
		{
			//next_iが第i方向かどうかチェック
			if(direction_i[i] == 0)
			{
				//この方向には進まないので別の方向をチェック
				continue;
			}
			//toの位置に達していたらこれ以上は進まないので別の方向をチェック
			if(p_i[i] == to_i[i])
			{
				//この方向には進まないので別の方向をチェック
				continue;
			}
			//i方向と垂直な平面との交点を求める
			float cross[4];
			getCrossPoint(from, to, p_i, interval, direction_i, i, cross);
			//現在のセルp_iの境界上の点であればnext_iを確定してbreakでforを抜ける
			if(atBound(cross, p_i, interval, i))
			{
				next_i[i] += direction_i[i];
				break;
			}
		}
		
		//カウントアップを行う
		//対象のカウンタのインデックス
		int c_i = 0;//TODO カウンタのインデックスを算出する関数を作成する
		for(int i = 0; i < 4; i++)
		{
			int c_dir_i = next_i[i] - p_i[i];
			if(c_dir_i != 0)
			{
				c_i += 2 * i + ((c_dir_i > 0) ? 0 : 1);//向きに応じたカウンタのインデックス
				break;
			}
		}
		atomicAdd(&(c[c_i]),1);//カウンタにatomicに加算する <- __host__向けには使えない！！//TODO 修正する！
		
		//p_iをnext_iに更新
		p_i[0] = next_i[0];
		p_i[1] = next_i[1];
		p_i[2] = next_i[2];
		p_i[3] = next_i[3];
	}
	while(!equals(p_i, to_i));//toのセルに到達したら終了
}

 */
