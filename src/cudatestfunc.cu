/*
 * =====================================================================================
 *
 *       Filename:  cudatestfunc.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2016年05月17日 00時26分44秒
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */


#include <hip/hip_runtime.h>





__global__ void test0(float* d_a)
{
	int i = threadIdx.x;
	d_a[i] *= 2.0f;
}



void cudatestfunc(float* h_a, int n)
{
	float* d_a;
	hipMalloc((void**)&d_a, n * sizeof(float));
	hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
	test0<<<1,n>>>(d_a);
	hipMemcpy(h_a, d_a, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_a);
}
